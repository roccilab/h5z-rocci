#include "hip/hip_runtime.h"
#include <cuSZp/cuSZp_f32.h>
#include <iostream>
#include "qcat_ssim.h"

__global__ void estimate_SZp_compress_kernel_f32(const float* const __restrict__ oriData, double* total_compressed_size, const float eb, const size_t nbEle, const size_t sample_stride);
__global__ void estimate_SZp_psnr_kernel_f32(const float* const __restrict__ oriData, float* maxVals, float* minVals, double* square_errors, const float eb, const size_t nbEle, const size_t sample_stride);
__global__ void estimate_SZp_ssim_kernel_f32(const float* const __restrict__ oriData, size_t n_dim, size_t dims[3], float* sampleData, float* decData, const float eb, const size_t nbEle, const size_t sample_stride);


double SZp_estimate_compress_hostptr_f32(float* oriData, size_t nbEle, float errorBound, size_t sample_stride)
{
    // Data blocking.
    int bsize = cmp_tblock_size_f32;
    int gsize = (nbEle + bsize * cmp_chunk_f32 - 1) / (bsize * cmp_chunk_f32);
    int sample_gsize = ((nbEle / sample_stride) + bsize * cmp_chunk_f32 - 1) / (bsize * cmp_chunk_f32);
    int cmpOffSize = gsize + 1;
    int pad_nbEle = gsize * bsize * cmp_chunk_f32;
    if (pad_nbEle > nbEle){
        pad_nbEle = nbEle;
    }

    size_t BLOCKSIZE = 32;
    size_t nbBlocks = pad_nbEle / BLOCKSIZE;
    size_t nbChunks = pad_nbEle / cmp_chunk_f32;

    // Initializing global memory for GPU compression.
    float* d_oriData;
    double* d_total_compressed_size;
    hipMalloc((void**)&d_oriData, sizeof(float)*pad_nbEle);
    hipMemcpy(d_oriData, oriData, sizeof(float)*pad_nbEle, hipMemcpyHostToDevice);
    // hipMalloc(&d_total_compressed_size, sizeof(double) * nbBlocks);
    // hipMemset(d_total_compressed_size, 0, sizeof(double) * nbBlocks);
    hipMalloc(&d_total_compressed_size, sizeof(double) * nbChunks);
    hipMemset(d_total_compressed_size, 0, sizeof(double) * nbChunks);
    hipMemset(d_oriData + nbEle, 0, (pad_nbEle - nbEle) * sizeof(float));

    // Initializing CUDA Stream.
    hipStream_t stream;
    hipStreamCreate(&stream);

    // cuSZp GPU compression.
    dim3 blockSize(bsize);
    dim3 gridSize(sample_gsize); // only launch sample grid num blocks
    estimate_SZp_compress_kernel_f32<<<gridSize, blockSize, 0, stream>>>(d_oriData, d_total_compressed_size, errorBound, nbEle, sample_stride);
    hipDeviceSynchronize();

    // Obtain compression ratio and move data back to CPU. 
    // double* h_total_compressed_size = (double*) malloc(sizeof(double) * nbBlocks);
    // hipMemcpy(h_total_compressed_size, d_total_compressed_size, sizeof(double)*nbBlocks, hipMemcpyDeviceToHost); 
    double* h_total_compressed_size = (double*) malloc(sizeof(double) * nbChunks);
    hipMemcpy(h_total_compressed_size, d_total_compressed_size, sizeof(double)*nbChunks, hipMemcpyDeviceToHost); 


    // compute on chunk stride instead of block stride
    // size_t nbSampleBlocks = (nbBlocks / sample_stride);
    // size_t nbSampleEle = nbSampleBlocks * BLOCKSIZE;
    // double original_size = nbSampleEle * sizeof(float);
    // size_t nbChunk = nbEle / cmp_chunk_f32;
    size_t nbSampleChunk = nbChunks / sample_stride;
    size_t nbSampleEle = nbSampleChunk * cmp_chunk_f32;
    double original_size = nbSampleEle * sizeof(float);

    // sum chunk compressed sizes
    double total_size = 0;
    // for(int i = 0; i < nbBlocks; i+=sample_stride) {
    for(int i = 0; i < nbChunks; i+=sample_stride) {
        total_size += h_total_compressed_size[i];
    }

    double compression_ratio = original_size / (total_size);

    // Free memory that is used.
    hipFree(d_oriData);
    hipFree(d_total_compressed_size);
    free(h_total_compressed_size);
    hipStreamDestroy(stream);

    return compression_ratio;
}


double SZp_estimate_psnr_hostptr_f32(float* oriData, size_t nbEle, float errorBound, size_t sample_stride)
{
    // Data blocking.
    int bsize = cmp_tblock_size_f32;
    int gsize = (nbEle + bsize * cmp_chunk_f32 - 1) / (bsize * cmp_chunk_f32);
    int cmpOffSize = gsize + 1;
    int pad_nbEle = gsize * bsize * cmp_chunk_f32;
    if (pad_nbEle > nbEle){
        pad_nbEle = nbEle;
    }
    int num_chunks = pad_nbEle / cmp_chunk_f32;

    size_t BLOCKSIZE = 32;
    size_t nbBlocks = pad_nbEle / BLOCKSIZE;

    // Initializing global memory for GPU compression.
    float* d_oriData;
    float* d_maxVals;
    float* d_minVals;
    double* d_errors;
    hipMalloc((void**)&d_oriData, sizeof(float)*pad_nbEle);
    hipMemcpy(d_oriData, oriData, sizeof(float)*pad_nbEle, hipMemcpyHostToDevice);
    hipMalloc(&d_maxVals, sizeof(float) * num_chunks);
    hipMemset(d_maxVals, 0, sizeof(float) * num_chunks);
    hipMalloc(&d_minVals, sizeof(float) * num_chunks);
    hipMemset(d_minVals, 0, sizeof(float) * num_chunks);
    hipMalloc(&d_errors, sizeof(double) * num_chunks);
    hipMemset(d_errors, 0, sizeof(double) * num_chunks);
    hipMemset(d_oriData + nbEle, 0, (pad_nbEle - nbEle) * sizeof(float));

    // Initializing CUDA Stream.
    hipStream_t stream;
    hipStreamCreate(&stream);

    // cuSZp GPU compression.
    dim3 blockSize(bsize);
    dim3 gridSize(gsize);
    estimate_SZp_psnr_kernel_f32<<<gridSize, blockSize, 0, stream>>>(d_oriData, d_maxVals, d_minVals, d_errors, errorBound, nbEle, sample_stride);
    hipDeviceSynchronize();

    // Obtain compression ratio and move data back to CPU. 
    float* h_maxVals = (float*) malloc(sizeof(float) * num_chunks);
    hipMemcpy(h_maxVals, d_maxVals, sizeof(float)*num_chunks, hipMemcpyDeviceToHost); 
    float* h_minVals = (float*) malloc(sizeof(float) * num_chunks);
    hipMemcpy(h_minVals, d_minVals, sizeof(float)*num_chunks, hipMemcpyDeviceToHost); 
    double* h_errors = (double*) malloc(sizeof(double) * num_chunks);
    hipMemcpy(h_errors, d_errors, sizeof(double)*num_chunks, hipMemcpyDeviceToHost); 

    size_t nbSampleBlocks = (nbBlocks / sample_stride);
    size_t nbSampleEle = nbSampleBlocks * BLOCKSIZE;
    double original_size = nbSampleEle * sizeof(float);

    // aggregate values
    float maxVal = h_maxVals[0];
    float minVal = h_minVals[0];
    double sq_err = 0;
    for(int i = 0; i < num_chunks; i++) {
        if(h_maxVals[i] > maxVal) maxVal = h_maxVals[i];
        if(h_minVals[i] < minVal) minVal = h_minVals[i];
        sq_err += h_errors[i];
    }

    double valRange = maxVal - minVal;
    double mse = sq_err / nbSampleEle;
    double eps = 1e-16;
    double psnr = -20.0*log10((sqrt(mse) / valRange) + eps);

    // Free memory that is used.
    hipFree(d_oriData);
    hipFree(d_maxVals);
    hipFree(d_minVals);
    hipFree(d_errors);
    free(h_maxVals);
    free(h_minVals);
    free(h_errors);
    hipStreamDestroy(stream);

    return psnr;
}

double SZp_estimate_ssim_hostptr_f32(float* oriData, size_t dims[3], size_t n_dim, size_t nbEle, float errorBound, size_t sample_stride)
{
    // Data blocking.
    int bsize = cmp_tblock_size_f32;
    int gsize = (nbEle + bsize * cmp_chunk_f32 - 1) / (bsize * cmp_chunk_f32);
    int cmpOffSize = gsize + 1;
    int pad_nbEle = gsize * bsize * cmp_chunk_f32;
    if (pad_nbEle > nbEle){
        pad_nbEle = nbEle;
    }

    size_t BLOCKSIZE = 32;
    size_t nbBlocks = pad_nbEle / BLOCKSIZE;
    size_t nbSampleBlocks = (nbBlocks / sample_stride);
    size_t nbSampleEle = nbSampleBlocks * BLOCKSIZE;
    float sample_rate = 1.0 / sample_stride;

    size_t sample_dims[3];
    size_t num_sample_points = 1;
    // for(size_t i = 0; i < 3; i++){
    //     if(n_dim == 1){
    //         sample_dims[i] = dims[i] / sample_stride;
    //     }
    //     else if(n_dim == 2){
    //         sample_dims[i] = sqrt(sample_rate)*dims[i];
    //     }
    //     else {
    //         sample_dims[i] = std::cbrt(sample_rate)*dims[i];
    //     }
    //     if(sample_dims[i] > 0){
    //         num_sample_points *= sample_dims[i];
    //     }
    // }
    // nbSampleEle = num_sample_points;

    if(n_dim == 1){
        sample_dims[0] = dims[0] / (float) sample_stride;
    }
    else if(n_dim == 2){
        sample_dims[0] = dims[0] / (float) sample_stride;
        sample_dims[1] = dims[1];
    }
    else {
        sample_dims[0] = dims[0] / (float) sample_stride;
        sample_dims[1] = dims[1];
        sample_dims[2] = dims[2];
    }

    printf("Ndim %i, SS: %i, dims: %i %i %i\n", n_dim, sample_stride, sample_dims[0], sample_dims[1], sample_dims[2]);

    // Initializing global memory for GPU compression.
    float* d_oriData;
    float* d_sampleData;
    float* d_decData;
    hipMalloc((void**)&d_oriData, sizeof(float)*pad_nbEle);
    hipMemcpy(d_oriData, oriData, sizeof(float)*pad_nbEle, hipMemcpyHostToDevice);
    hipMalloc(&d_sampleData, sizeof(float) * nbSampleEle);
    hipMemset(d_sampleData, 0, sizeof(float) * nbSampleEle);
    hipMalloc(&d_decData, sizeof(float) * nbSampleEle);
    hipMemset(d_decData, 0, sizeof(float) * nbSampleEle);
    hipMemset(d_oriData + nbEle, 0, (pad_nbEle - nbEle) * sizeof(float));

    // Initializing CUDA Stream.
    hipStream_t stream;
    hipStreamCreate(&stream);

    // cuSZp GPU compression.
    dim3 blockSize(bsize);
    dim3 gridSize(gsize);
    estimate_SZp_ssim_kernel_f32<<<gridSize, blockSize, 0, stream>>>(d_oriData, n_dim, dims, d_sampleData, d_decData, errorBound, nbEle, sample_stride);
    hipDeviceSynchronize();

    // Obtain compression ratio and move data back to CPU. 
    float* h_sampleData = (float*) malloc(sizeof(float) * nbSampleEle);
    hipMemcpy(h_sampleData, d_sampleData, sizeof(float)*nbSampleEle, hipMemcpyDeviceToHost); 
    float* h_decData = (float*) malloc(sizeof(float) * nbSampleEle);
    hipMemcpy(h_decData, d_decData, sizeof(float)*nbSampleEle, hipMemcpyDeviceToHost); 

    double ssim;
	if(n_dim == 1){
		ssim = SSIM_1d_windowed_float(h_sampleData, h_decData, sample_dims[0], 8, 8);
	}
	else if (n_dim == 2) {
		ssim = SSIM_2d_windowed_float(h_sampleData, h_decData, sample_dims[1], sample_dims[0], 8,8, 8,8);
	}
	else {
		ssim = SSIM_3d_windowed_float(h_sampleData, h_decData, sample_dims[2], sample_dims[1], sample_dims[0], 8,8,8, 8,8,8);
	}

    // Free memory that is used.
    hipFree(d_oriData);
    hipFree(d_sampleData);
    hipFree(d_decData);
    free(h_sampleData);
    free(h_decData);
    hipStreamDestroy(stream);

    return ssim;
}
