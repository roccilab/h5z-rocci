#include "hip/hip_runtime.h"
#include <cuSZp/cuSZp_f32.h>

__device__ inline int quantization_f32(float data, float recipPrecision)
{
    float dataRecip = data*recipPrecision;
    int s = dataRecip>=-0.5f?0:1;
    return (int)(dataRecip+0.5f) - s;
}


__device__ inline int get_bit_num(unsigned int x)
{
    return (sizeof(unsigned int)*8) - __clz(x);
}

// CR
__global__ void estimate_SZp_compress_kernel_f32(const float* const __restrict__ oriData, double* total_compressed_size, const float eb, const size_t nbEle, const size_t sample_stride)
{
    __shared__ unsigned int base_idx;

    const int tid = threadIdx.x;
    // scale by stride
    const int idx = (blockIdx.x * blockDim.x + tid) * sample_stride;
    const int lane = idx & 31;
    const int warp = idx >> 5;
    const int block_num = cmp_chunk_f32/32;
    const int start_idx = idx * cmp_chunk_f32;
    const int start_block_idx = start_idx/32;
    const int rate_ofs = (nbEle+31)/32;
    const float recipPrecision = 0.5f/eb;

    int temp_start_idx, temp_end_idx;
    int quant_chunk_idx;
    int block_idx;
    int currQuant, lorenQuant, prevQuant, maxQuant;
    int absQuant[cmp_chunk_f32];
    unsigned int thread_ofs = 0;

    size_t num_zero_blocks = 0;
    double local_compressed_size = 0;

    // changes to chunk stride
    if(start_idx + cmp_chunk_f32 < nbEle){

        // iterate by sample stride to skip blocks
        // for(int j=0; j<block_num; j+=sample_stride)
        for(int j=0; j<block_num; j++)
        {
            temp_start_idx = start_idx + j*32;
            temp_end_idx = temp_start_idx + 32;
            block_idx = start_block_idx+j;

            double compressed_size = 0;

            prevQuant = 0;
            maxQuant = 0;
            
            bool is_zero_block = true;
            bool is_zero_quant_block = true;
            for(int i=temp_start_idx; i<temp_end_idx; i++){
                // compute lorenzo quantization index
                float curr = oriData[i];
                if(curr != 0) is_zero_block = false;
                currQuant = quantization_f32(curr, recipPrecision);
                if(currQuant != 0) is_zero_quant_block = false;
                lorenQuant = currQuant - prevQuant;
                prevQuant = currQuant;
                int absQuant = abs(lorenQuant);
                // update max required bit size for this block
                maxQuant = maxQuant > absQuant ? maxQuant : absQuant;   
            }

            // compute compressed size of this block in bytes
            if(is_zero_block || is_zero_quant_block){
                num_zero_blocks++;
                compressed_size += 1;
            }
            else {
                int block_bit_size = get_bit_num(maxQuant);
                compressed_size += (block_bit_size + 1) * 32 / 8;
            }

            // write out compressed size for this block
            // total_compressed_size[block_idx] = compressed_size;
            local_compressed_size += compressed_size;

        }

        total_compressed_size[idx] = local_compressed_size;
    }

}

// PSNR
__global__ void estimate_SZp_psnr_kernel_f32(const float* const __restrict__ oriData, float* max_vals, float* min_vals, double* square_errors, const float eb, const size_t nbEle, const size_t sample_stride)
{
    __shared__ unsigned int base_idx;

    const int tid = threadIdx.x;
    const int idx = blockIdx.x * blockDim.x + tid;
    const int lane = idx & 31;
    const int warp = idx >> 5;
    const int block_num = cmp_chunk_f32/32;
    const int start_idx = idx * cmp_chunk_f32;
    const int start_block_idx = start_idx/32;
    const int rate_ofs = (nbEle+31)/32;
    const float recipPrecision = 0.5f/eb;

    int temp_start_idx, temp_end_idx;
    int quant_chunk_idx;
    int block_idx;
    int currQuant, lorenQuant, prevQuant, maxQuant;
    int absQuant[cmp_chunk_f32];
    unsigned int thread_ofs = 0;

    size_t num_zero_blocks = 0;

    double sq_err = 0;
    float minVal = oriData[start_idx];
    float maxVal = oriData[start_idx];

    // iterate by sample stride to skip blocks
    for(int j=0; j<block_num; j+=sample_stride)
    {
        temp_start_idx = start_idx + j*32;
        temp_end_idx = temp_start_idx + 32;
        block_idx = start_block_idx+j;

        double compressed_size = 0;

        prevQuant = 0;
        maxQuant = 0;
        
        bool is_zero_block = true;
        bool is_zero_quant_block = true;
        for(int i=temp_start_idx; i<temp_end_idx; i++){
            // compute lorenzo quantization index
            float curr = oriData[i];
			if(curr < minVal) minVal = curr;
			if(curr > maxVal) maxVal = curr;

            currQuant = quantization_f32(curr, recipPrecision);
			float pointEstimate = currQuant * 2 * eb;
			sq_err += pow(curr - pointEstimate, 2);   
        }

    }

    max_vals[idx] = maxVal;
    min_vals[idx] = minVal;
    square_errors[idx] = sq_err;

}

__device__ size_t get_sampleidx(size_t idx, size_t sample_stride, size_t n_dim) 
{
    size_t sample_idx = idx;
    float sample_rate = 1.0 / sample_stride;
    if(n_dim == 1){
        sample_idx = sample_idx * sample_rate;
    }
    else if(n_dim == 2){
        sample_idx = sample_idx * sqrt(sample_rate);
    }
    else {
        sample_idx = sample_idx * std::cbrt(sample_rate);
    }
    return sample_idx;
}

// SSIM
// __global__ void estimate_SZp_ssim_kernel_f32(const float* const __restrict__ oriData, size_t n_dim, float* sampleData, float* decData, const float eb, const size_t nbEle, const size_t sample_stride)
// {
//     __shared__ unsigned int base_idx;

//     const int tid = threadIdx.x;
//     const int idx = blockIdx.x * blockDim.x + tid;
//     const int lane = idx & 31;
//     const int warp = idx >> 5;
//     const int block_num = cmp_chunk_f32/32;
//     const int start_idx = idx * cmp_chunk_f32;
//     const int start_block_idx = start_idx/32;
//     const int rate_ofs = (nbEle+31)/32;
//     const float recipPrecision = 0.5f/eb;

//     size_t nbBlocks = nbEle / 32;
//     size_t nbSampleBlocks = (nbBlocks / sample_stride);
//     size_t nbSampleEle = nbSampleBlocks * 32;

//     int temp_start_idx, temp_end_idx;
//     int quant_chunk_idx;
//     int block_idx;
//     int currQuant, lorenQuant, prevQuant, maxQuant;
//     int absQuant[cmp_chunk_f32];
//     unsigned int thread_ofs = 0;

//     size_t num_zero_blocks = 0;
//     // set proper start position in sample array
//     // size_t sample_idx = start_idx;
//     // float sample_rate = 1.0 / sample_stride;
//     // if(n_dim == 1){
//     //     sample_idx = sample_idx * sample_rate;
//     // }
//     // else if(n_dim == 2){
//     //     sample_idx = sample_idx * sqrt(sample_rate);
//     // }
//     // else {
//     //     sample_idx = sample_idx * std::cbrt(sample_rate);
//     // }

//     // if(sample_idx < nbSampleEle){

//         // iterate by sample stride to skip blocks
//         for(int j=0; j<block_num; j+=sample_stride)
//         {
//             temp_start_idx = start_idx + j*32;
//             temp_end_idx = temp_start_idx + 32;
//             block_idx = start_block_idx+j;

//             size_t sample_idx = get_sampleidx(temp_start_idx, sample_stride, n_dim);

//             double compressed_size = 0;

//             prevQuant = 0;
//             maxQuant = 0;
            
//             bool is_zero_block = true;
//             bool is_zero_quant_block = true;
//             for(int i=temp_start_idx; i<temp_end_idx; i++){
                
//                 // compute lorenzo quantization index
//                 float curr = oriData[i];
//                 currQuant = quantization_f32(curr, recipPrecision);
//                 float pointEstimate = currQuant * 2 * eb;
                
//                 sampleData[sample_idx] = curr;
//                 decData[sample_idx] = pointEstimate;
//                 sample_idx++;

//                 if(sample_idx >= nbSampleEle) break;
//             }

//             // if(sample_idx >= nbSampleEle) break;

//         }
//     // }

// }

// __global__ void estimate_SZp_ssim_kernel_f32(const float* oriData, size_t n_dim, float* sampleData, float* decData, 
//                                              const float eb, const size_t nbEle, const size_t sample_stride)
// {
//     const int tid = blockIdx.x * blockDim.x + threadIdx.x;
//     const float recipPrecision = 0.5f / eb;
//     size_t nbBlocks = nbEle / 32;
//     size_t nbSampleBlocks = (nbBlocks / sample_stride);
//     size_t nbSampleEle = nbSampleBlocks * 32;

//     for (size_t i = tid; i < nbEle; i += gridDim.x * blockDim.x) {
//         if (i % (32 * sample_stride) < 32) {  // Only process every sample_stride'th block
//             size_t sample_idx = i / (32 * sample_stride) * 32 + (i % 32);
            
//             if (sample_idx < nbSampleEle) {
//                 float curr = oriData[i];
//                 int currQuant = quantization_f32(curr, recipPrecision);
//                 float pointEstimate = currQuant * 2 * eb;
                
//                 sampleData[sample_idx] = curr;
//                 decData[sample_idx] = pointEstimate;
//             }
//         }
//     }
// }

__device__ size_t multidim_sample_idx(size_t idx, size_t n_dim, size_t dims[3], size_t sample_dims[3], size_t sample_stride) {
    float sample_rate = 1 / sample_stride;
    if (n_dim == 1){
        return idx / sample_stride;
    }
    else if(n_dim == 2){
        size_t x = idx % dims[0];
        size_t y = idx / dims[1];
        size_t sample_x = x / (1/sqrt(sample_rate));
        size_t sample_y = y / (1/sqrt(sample_rate));
        return sample_x + sample_dims[0] * sample_y;
    }
    else {
        size_t x = idx % dims[0];
        size_t y = (idx / dims[0]) % dims[1];
        size_t z = idx / (dims[0]*dims[1]);
        size_t sample_x = x / (1/std::cbrt(sample_rate));
        size_t sample_y = y / (1/std::cbrt(sample_rate));
        size_t sample_z = z / (1/std::cbrt(sample_rate));
        return sample_x + sample_dims[0] * sample_y + sample_dims[1] * sample_z;
    }
}

__global__ void estimate_SZp_ssim_kernel_f32(const float* oriData, size_t n_dim, size_t dims[3], float* sampleData, float* decData, 
                                             const float eb, const size_t nbEle, const size_t sample_stride)
{
    const int tid = threadIdx.x;
    const int idx = blockIdx.x * blockDim.x + tid;
    const float recipPrecision = 0.5f / eb;
    const int block_num = cmp_chunk_f32/32;
    const int start_idx = idx * cmp_chunk_f32;
    const int start_block_idx = start_idx/32;
    size_t nbBlocks = nbEle / 32;
    size_t nbSampleBlocks = (nbBlocks / sample_stride);
    size_t nbSampleEle = nbSampleBlocks * 32;
    int currQuant, lorenQuant, prevQuant, maxQuant;
    int temp_start_idx, temp_end_idx;
    int quant_chunk_idx;
    int block_idx;

    size_t sample_dims[3];
    size_t num_sample_points = 1;
    float sample_rate = 1/sample_stride;
    // for(size_t i = 0; i < 3; i++){
    //     if(n_dim == 1){
    //         sample_dims[i] = dims[i] / sample_stride;
    //     }
    //     else if(n_dim == 2){
    //         sample_dims[i] = sqrt(sample_rate)*dims[i];
    //     }
    //     else {
    //         sample_dims[i] = std::cbrt(sample_rate)*dims[i];
    //     }
    //     if(sample_dims[i] > 0){
    //         num_sample_points *= sample_dims[i];
    //     }
    // }

    for (int j=0; j<block_num; j+=sample_stride) {
        temp_start_idx = start_idx + j*32;
        temp_end_idx = temp_start_idx + 32;
        block_idx = start_block_idx+j;
        for(int i=temp_start_idx; i<temp_end_idx; i++){
            // compute lorenzo quantization index
            float curr = oriData[i];
            currQuant = quantization_f32(curr, recipPrecision);
			float pointEstimate = currQuant * 2 * eb;

            size_t sample_idx = i / (32 * sample_stride) * 32 + (i % 32);
            // size_t sample_idx = multidim_sample_idx(i, n_dim, dims, sample_dims, sample_stride);
            if (sample_idx < nbSampleEle) {
                sampleData[sample_idx] = curr;
                decData[sample_idx] = pointEstimate;
            }

        }
    }
}